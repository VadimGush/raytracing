#include "hip/hip_runtime.h"
#include <iostream>
#include <exception>
#include <fstream>
#include <vector>
#include <glm/vec3.hpp>
#include <hiprand.h>
#include "utils/cuda_memory.h"
#include "utils/logger.h"
#include "utils/png.h"
#include "Display.h"
#include "RayTracer.h"
#include "Sphere.h"

using namespace std;
using namespace glm;

constexpr int display_width = 1920;
constexpr int display_height = 1080;

ostream& operator<<(ostream& output, const vec3& vector) {
    return output << vector.x << " " << vector.y << " " << vector.z;
}

CUDA::unique_ptr<float> GenerateRandomNumbers(const size_t size) {
    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, 123);
    CUDA::unique_ptr<float> device_random_numbers(size);
    hiprandGenerateUniform(
            generator,
            device_random_numbers.get_device_pointer().get(),
            device_random_numbers.get_device_pointer().size());
    return move(device_random_numbers);
}

CUDA::unique_ptr<Sphere> CreateWorld() {
    vector<Sphere> spheres = {
            Sphere{ 0.3,   { 0,     0, -1}      , Material::Scatter({1,1,1})}, // big red sphere
            Sphere{ 20,    { 0, -20.30, -1}     , Material::Scatter({0.1,1,0.1})}, // floor
            Sphere{ 0.2,   { 0.6, 0, -1.1}      , Material::Metal({1,1,0}, 0.1)},
            Sphere{ 0.06,  { 0.35,-0.2, -1.1}   , Material::Light({1,1,1})},
            Sphere{ 0.02,  { 0.35,-0.25, -0.8}  , Material::Light({1,0,1})},
            Sphere{ 0.05,  { 0.25,-0.25, -0.9}  , Material::Metal({1,1,1}, 1)},
            Sphere{ 0.04,  { 0.15,-0.25, -0.8}  , Material::Light({0,1,1})},
            Sphere{ 0.1,   { 0.5, -0.3, -0.9}   , Material::Metal({1,1,1}, 0.9)},
            Sphere{ 0.04,  { 0.3, -0.27, -0.7}   , Material::Dielectric(1.5f)},
    };
    CUDA::unique_ptr<Sphere> device_spheres(spheres.size());
    device_spheres.copy_from(spheres.data());
    return move(device_spheres);
}

int main() {

    try {
        Display display{display_width, display_height};

        const auto world = CreateWorld();
        const auto random_numbers = GenerateRandomNumbers(display_width * display_height);

        dim3 threads(16, 16);
        dim3 blocks(display_width / threads.x + 1, display_height / threads.y + 1);

        Logger::info() << "Rendering" << endl;

        // RENDER
        RayTracer::RenderScreen<<<blocks, threads>>>(
                world.get_device_pointer(),
                random_numbers.get_device_pointer(),
                display.GetDisplay(), display_width, display_height
        );
        const auto image = display.GetImage();

        Logger::info() << "Exporting image to PNG" << endl;
        PNG::WriteImage("output.png", display_width, display_height, image);

        Logger::info() << "Done!" << endl;
    } catch (const exception& e) {
        Logger::fatal() << e.what() << endl;
    }

    return 0;
}
