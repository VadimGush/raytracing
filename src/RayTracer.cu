#include "hip/hip_runtime.h"
//
// Created by Vadim Gush on 27.12.2019.
//

#include "RayTracer.cuh"

#include <glm/geometric.hpp>
#include <cstdlib>

#define RAY_COUNT 50

using namespace glm;

class Rand {
public:
    __device__ Rand(const int);

    // generate real number in [0, 1]
    __device__ float Float();

    // generate real number in [-1, 1]
    __device__ float FullFloat();
private:
    int seed_;
};

__device__ unsigned long xorshf96(int value) {
    unsigned long x=123456789 + value, y=362436069, z=521288629;
    unsigned long t;
    x ^= x << 16;
    x ^= x >> 5;
    x ^= x << 1;
    t = x;
    x = y;
    y = z;
    z = t ^ x ^ y;
    return z;
}

__device__ Rand::Rand(const int seed) : seed_(seed) {}

__device__ float Rand::Float() {
    seed_++;
    return static_cast<float>(xorshf96(seed_) % 10000) / 10000.0f;
}

__device__ float Rand::FullFloat() {
    return Float() * 2 - 1;
}

struct Hit {
    float distance;
    vec3 color;
    vec3 normal;
};

__device__ Hit HitSphere(const vec3& center, const float radius, const vec3& origin, const vec3& direction) {
    vec3 oc = origin - center;
    float a = dot(direction, direction);
    float b = 2 * dot(oc, direction);
    float c = dot(oc, oc) - radius * radius;
    float d = b * b - 4 * a * c;
    if (d < 0) {
        return {-1, {}, {}};
    } else {
        float t = (-b - sqrt(d)) / (2*a);
        vec3 hit_position = origin + direction * t;
        vec3 sphere_normal = hit_position - center;

        return {t, {1,radius,1}, normalize(sphere_normal)};
    }
}

__device__ vec3 Render(Sphere* spheres, const int spheres_count, const vec3& camera_origin, const vec3& camera_direction, int iter, Rand& rand) {
    vec3 sphere_center = vec3{0, 0, -1};

    Hit hit{-1, {0,0,0}, {0,0,0}};
    for (int i = 0; i < spheres_count; i++) {
        Sphere& sphere = spheres[i];
        Hit current = HitSphere(sphere.position, sphere.radius, camera_origin, camera_direction);

        if (hit.distance <= 0) {
            hit = current;
        } else if (hit.distance > current.distance && current.distance > 0) {
            hit = current;
        }
    }

    if (hit.distance > 0.0001) {

        vec3 random{rand.Float()*2-1, rand.Float()*2-1, rand.Float()*2-1};
        vec3 point = camera_origin + camera_direction * hit.distance;

        iter++;
        if (iter < 6)
            return 0.5f * Render(spheres, spheres_count, point, normalize(hit.normal + random), iter, rand);
        else
            return {0,0,0};
    } else {
        return {0.5,0.5,1};
    }
}

__global__ void RayTracer::RenderScreen(
        cuda_device_ptr<Sphere> spheres,
        const int spheres_count,
        cuda_device_ptr<vec3> display_ptr,
        const int display_width,
        const int display_height) {

    vec3* display = display_ptr.get();
    int xi = threadIdx.x + blockDim.x * blockIdx.x;
    int yi = threadIdx.y + blockDim.y * blockIdx.y;
    vec3& pixel = display[xi + yi * display_width];

    if (xi < display_width && yi < display_height) {

        Rand rand((xi + display_width * yi) * 100);

        vec3 color{0,0,0};
        for (int i = 0; i < RAY_COUNT; i++) {

            float x = ((float)xi + rand.Float()) / display_width - 0.5;
            float y = ((float)yi + rand.Float()) / display_height - 0.5;
            float aspect = (float) display_width / display_height;
            x *= aspect;

            color += Render(spheres.get(), spheres_count, {0,0,0}, {x, y, -1}, 0, rand);
        }
        pixel = color / (float)RAY_COUNT;

        // debug
        // if (threadIdx.x == 0 || threadIdx.y == 0) pixel.b += 0.5;
        // if (x == 0 || y == 0) pixel.r += 0.5;
    }
}
